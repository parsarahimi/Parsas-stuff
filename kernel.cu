#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "gputimer.h"
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>

#define numElements 20480
__device__ float3
tile_calculation(float3 accel)
{
	extern volatile __shared__ float4 shPosition[];
	int b = 1;//threadIdx.x;
	accel.x += shPosition[b].x;//problem is here in this function pretty much, where we access shared mem
	return accel;
}
__global__ void
__launch_bounds__(1024, 2)
calculate_forces(void *devX, void *devA)
{
	extern volatile __shared__ float4 shPosition[];
	float4 *globalX = (float4 *)devX;
	float4 *globalA = (float4 *)devA;
	float3 acc = { 0.0f, 0.0f, 0.0f };
	int section;
	int gtid = blockIdx.x * blockDim.x + threadIdx.x;
	for (section = 0; section < 20; section++) {
		int idx = section * blockDim.x + threadIdx.x;
		const_cast<float4&>(shPosition[threadIdx.x]) = globalX[idx];
		__syncthreads();
		acc = tile_calculation(acc);// comment out this line, which has to do with reading the shared memory
		__syncthreads();//and see the performance difference!
	}
	// Save the result in global memory.
	float4 acc4 = { acc.x, acc.y, acc.z, 0.0f };
	globalA[gtid] = acc4;
}
int main()
{
	size_t size = 4 * numElements * sizeof(float);
	printf("[Vector addition of %d elements]\n", numElements);
	// Allocate the host input vectors pos and force.
	float4 *h_pos = (float4 *)malloc(size);
	float4 *h_force = (float4 *)malloc(size);

	// Initialize the host input vectors
	for (int i = 0; i < numElements; ++i)
	{
		h_pos[i].x = 10 * rand() / (float)RAND_MAX;//Generate a random number between 0 and 10
		h_pos[i].y = 10 * rand() / (float)RAND_MAX;
		h_pos[i].z = 10 * rand() / (float)RAND_MAX;
		h_pos[i].w = rand() / (float)RAND_MAX;//Generate a random number between 0 and 1
	}

	hipError_t err = hipSuccess;
	float4 *d_pos = NULL;
	err = hipMalloc((void **)&d_pos, size);
	float4 *d_force = NULL;
	err = hipMalloc((void **)&d_force, size);

	int threadsPerBlock = 1024;
	int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;

	printf("%d\n", blocksPerGrid);
	printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
	GpuTimer timer;
	timer.Start();
	calculate_forces<<<blocksPerGrid, threadsPerBlock, 4 * 1024 * sizeof(float) >>> (d_pos, d_force);//
	timer.Stop();
	printf("Time took: %g ms\n", timer.Elapsed());

	printf("Done\n");
    return 0;
}
